#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "stdint.h"
#include <stddef.h>
#include <stdarg.h>
#include<string.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include <sys/timeb.h>

__managed__ int found = 0;
__managed__ char* word = "cata";
__managed__ char* hash = "a31ae9fe898b3f1d73e28d0d501014e3385ac1d4";
__managed__ char result[20];

__device__ static void simple_outputchar(char** str, char c)
{
	if (str) {
		**str = c;
		++(*str);
	}
	else {
		//putchar(c);
	}
}

enum flags {
	PAD_ZERO = 1,
	PAD_RIGHT = 2,
};

__device__ static int prints(char** out, const char* string, int width, int flags)
{
	int pc = 0, padchar = ' ';

	if (width > 0) {
		int len = 0;
		const char* ptr;
		for (ptr = string; *ptr; ++ptr) ++len;
		if (len >= width) width = 0;
		else width -= len;
		if (flags & PAD_ZERO)
			padchar = '0';
	}
	if (!(flags & PAD_RIGHT)) {
		for (; width > 0; --width) {
			simple_outputchar(out, padchar);
			++pc;
		}
	}
	for (; *string; ++string) {
		simple_outputchar(out, *string);
		++pc;
	}
	for (; width > 0; --width) {
		simple_outputchar(out, padchar);
		++pc;
	}

	return pc;
}

#define PRINT_BUF_LEN 64

__device__ static int simple_outputi(char** out, long long i, int base, int sign, int width, int flags, int letbase)
{
	char print_buf[PRINT_BUF_LEN];
	char* s;
	int t, neg = 0, pc = 0;
	unsigned long long u = i;

	if (i == 0) {
		print_buf[0] = '0';
		print_buf[1] = '\0';
		return prints(out, print_buf, width, flags);
	}

	if (sign && base == 10 && i < 0) {
		neg = 1;
		u = -i;
	}

	s = print_buf + PRINT_BUF_LEN - 1;
	*s = '\0';

	while (u) {
		t = u % base;
		if (t >= 10)
			t += letbase - '0' - 10;
		*--s = t + '0';
		u /= base;
	}

	if (neg) {
		if (width && (flags & PAD_ZERO)) {
			simple_outputchar(out, '-');
			++pc;
			--width;
		}
		else {
			*--s = '-';
		}
	}

	return pc + prints(out, s, width, flags);
}


__device__ static int simple_vsprintf(char** out, char* format, va_list ap)
{
	int width, flags;
	int pc = 0;
	char scr[2];
	union {
		char c;
		char* s;
		int i;
		unsigned int u;
		long li;
		unsigned long lu;
		long long lli;
		unsigned long long llu;
		short hi;
		unsigned short hu;
		signed char hhi;
		unsigned char hhu;
		void* p;
	} u;

	for (; *format != 0; ++format) {
		if (*format == '%') {
			++format;
			width = flags = 0;
			if (*format == '\0')
				break;
			if (*format == '%')
				goto out;
			if (*format == '-') {
				++format;
				flags = PAD_RIGHT;
			}
			while (*format == '0') {
				++format;
				flags |= PAD_ZERO;
			}
			if (*format == '*') {
				width = va_arg(ap, int);
				format++;
			}
			else {
				for (; *format >= '0' && *format <= '9'; ++format) {
					width *= 10;
					width += *format - '0';
				}
			}
			switch (*format) {
			case('d'):
				u.i = va_arg(ap, int);
				pc += simple_outputi(out, u.i, 10, 1, width, flags, 'a');
				break;

			case('u'):
				u.u = va_arg(ap, unsigned int);
				pc += simple_outputi(out, u.u, 10, 0, width, flags, 'a');
				break;

			case('x'):
				u.u = va_arg(ap, unsigned int);
				pc += simple_outputi(out, u.u, 16, 0, width, flags, 'a');
				break;

			case('X'):
				u.u = va_arg(ap, unsigned int);
				pc += simple_outputi(out, u.u, 16, 0, width, flags, 'A');
				break;

			case('c'):
				u.c = va_arg(ap, int);
				scr[0] = u.c;
				scr[1] = '\0';
				pc += prints(out, scr, width, flags);
				break;

			case('s'):
				u.s = va_arg(ap, char*);
				pc += prints(out, u.s ? u.s : "(null)", width, flags);
				break;
			case('l'):
				++format;
				switch (*format) {
				case('d'):
					u.li = va_arg(ap, long);
					pc += simple_outputi(out, u.li, 10, 1, width, flags, 'a');
					break;

				case('u'):
					u.lu = va_arg(ap, unsigned long);
					pc += simple_outputi(out, u.lu, 10, 0, width, flags, 'a');
					break;

				case('x'):
					u.lu = va_arg(ap, unsigned long);
					pc += simple_outputi(out, u.lu, 16, 0, width, flags, 'a');
					break;

				case('X'):
					u.lu = va_arg(ap, unsigned long);
					pc += simple_outputi(out, u.lu, 16, 0, width, flags, 'A');
					break;

				case('l'):
					++format;
					switch (*format) {
					case('d'):
						u.lli = va_arg(ap, long long);
						pc += simple_outputi(out, u.lli, 10, 1, width, flags, 'a');
						break;

					case('u'):
						u.llu = va_arg(ap, unsigned long long);
						pc += simple_outputi(out, u.llu, 10, 0, width, flags, 'a');
						break;

					case('x'):
						u.llu = va_arg(ap, unsigned long long);
						pc += simple_outputi(out, u.llu, 16, 0, width, flags, 'a');
						break;

					case('X'):
						u.llu = va_arg(ap, unsigned long long);
						pc += simple_outputi(out, u.llu, 16, 0, width, flags, 'A');
						break;

					default:
						break;
					}
					break;
				default:
					break;
				}
				break;
			case('h'):
				++format;
				switch (*format) {
				case('d'):
					u.hi = va_arg(ap, int);
					pc += simple_outputi(out, u.hi, 10, 1, width, flags, 'a');
					break;

				case('u'):
					u.hu = va_arg(ap, unsigned int);
					pc += simple_outputi(out, u.lli, 10, 0, width, flags, 'a');
					break;

				case('x'):
					u.hu = va_arg(ap, unsigned int);
					pc += simple_outputi(out, u.lli, 16, 0, width, flags, 'a');
					break;

				case('X'):
					u.hu = va_arg(ap, unsigned int);
					pc += simple_outputi(out, u.lli, 16, 0, width, flags, 'A');
					break;

				case('h'):
					++format;
					switch (*format) {
					case('d'):
						u.hhi = va_arg(ap, int);
						pc += simple_outputi(out, u.hhi, 10, 1, width, flags, 'a');
						break;

					case('u'):
						u.hhu = va_arg(ap, unsigned int);
						pc += simple_outputi(out, u.lli, 10, 0, width, flags, 'a');
						break;

					case('x'):
						u.hhu = va_arg(ap, unsigned int);
						pc += simple_outputi(out, u.lli, 16, 0, width, flags, 'a');
						break;

					case('X'):
						u.hhu = va_arg(ap, unsigned int);
						pc += simple_outputi(out, u.lli, 16, 0, width, flags, 'A');
						break;

					default:
						break;
					}
					break;
				default:
					break;
				}
				break;
			default:
				break;
			}
		}
		else {
		out:
			simple_outputchar(out, *format);
			++pc;
		}
	}
	if (out) **out = '\0';
	return pc;
}

__device__ int simple_printf(char* fmt, ...)
{
	va_list ap;
	int r;

	va_start(ap, fmt);
	r = simple_vsprintf(NULL, fmt, ap);
	va_end(ap);

	return r;
}

__device__ int simple_sprintf(char* buf, char* fmt, ...)
{
	va_list ap;
	int r;

	va_start(ap, fmt);
	r = simple_vsprintf(&buf, fmt, ap);
	va_end(ap);

	return r;
}

__device__ int sha1digest(uint8_t* digest, char* hexdigest, const uint8_t* data, size_t databytes) {
#define SHA1ROTATELEFT(value, bits) (((value) << (bits)) | ((value) >> (32 - (bits))))

	uint32_t W[80];
	uint32_t H[] = { 0x67452301,
		0xEFCDAB89,
		0x98BADCFE,
		0x10325476,
		0xC3D2E1F0 };
	uint32_t a;
	uint32_t b;
	uint32_t c;
	uint32_t d;
	uint32_t e;
	uint32_t f = 0;
	uint32_t k = 0;

	uint32_t idx;
	uint32_t lidx;
	uint32_t widx;
	uint32_t didx = 0;

	int32_t wcount;
	uint32_t temp;
	uint64_t databits = ((uint64_t)databytes) * 8;
	uint32_t loopcount = (databytes + 8) / 64 + 1;
	uint32_t tailbytes = 64 * loopcount - databytes;
	uint8_t datatail[128] = { 0 };

	if (!digest && !hexdigest)
		return -1;

	if (!data)
		return -1;

	/* Pre-processing of data tail (includes padding to fill out 512-bit chunk):
	Add bit '1' to end of message (big-endian)
	Add 64-bit message length in bits at very end (big-endian) */
	datatail[0] = 0x80;
	datatail[tailbytes - 8] = (uint8_t)(databits >> 56 & 0xFF);
	datatail[tailbytes - 7] = (uint8_t)(databits >> 48 & 0xFF);
	datatail[tailbytes - 6] = (uint8_t)(databits >> 40 & 0xFF);
	datatail[tailbytes - 5] = (uint8_t)(databits >> 32 & 0xFF);
	datatail[tailbytes - 4] = (uint8_t)(databits >> 24 & 0xFF);
	datatail[tailbytes - 3] = (uint8_t)(databits >> 16 & 0xFF);
	datatail[tailbytes - 2] = (uint8_t)(databits >> 8 & 0xFF);
	datatail[tailbytes - 1] = (uint8_t)(databits >> 0 & 0xFF);

	/* Process each 512-bit chunk */
	for (lidx = 0; lidx < loopcount; lidx++)
	{
		/* Compute all elements in W */
		memset(W, 0, 80 * sizeof(uint32_t));

		/* Break 512-bit chunk into sixteen 32-bit, big endian words */
		for (widx = 0; widx <= 15; widx++)
		{
			wcount = 24;

			/* Copy byte-per byte from specified buffer */
			while (didx < databytes && wcount >= 0)
			{
				W[widx] += (((uint32_t)data[didx]) << wcount);
				didx++;
				wcount -= 8;
			}
			/* Fill out W with padding as needed */
			while (wcount >= 0)
			{
				W[widx] += (((uint32_t)datatail[didx - databytes]) << wcount);
				didx++;
				wcount -= 8;
			}
		}

		/* Extend the sixteen 32-bit words into eighty 32-bit words, with potential optimization from:
		"Improving the Performance of the Secure Hash Algorithm (SHA-1)" by Max Locktyukhin */
		for (widx = 16; widx <= 31; widx++)
		{
			W[widx] = SHA1ROTATELEFT((W[widx - 3] ^ W[widx - 8] ^ W[widx - 14] ^ W[widx - 16]), 1);
		}
		for (widx = 32; widx <= 79; widx++)
		{
			W[widx] = SHA1ROTATELEFT((W[widx - 6] ^ W[widx - 16] ^ W[widx - 28] ^ W[widx - 32]), 2);
		}

		/* Main loop */
		a = H[0];
		b = H[1];
		c = H[2];
		d = H[3];
		e = H[4];

		for (idx = 0; idx <= 79; idx++)
		{
			if (idx <= 19)
			{
				f = (b & c) | ((~b) & d);
				k = 0x5A827999;
			}
			else if (idx >= 20 && idx <= 39)
			{
				f = b ^ c ^ d;
				k = 0x6ED9EBA1;
			}
			else if (idx >= 40 && idx <= 59)
			{
				f = (b & c) | (b & d) | (c & d);
				k = 0x8F1BBCDC;
			}
			else if (idx >= 60 && idx <= 79)
			{
				f = b ^ c ^ d;
				k = 0xCA62C1D6;
			}
			temp = SHA1ROTATELEFT(a, 5) + f + e + k + W[idx];
			e = d;
			d = c;
			c = SHA1ROTATELEFT(b, 30);
			b = a;
			a = temp;
		}

		H[0] += a;
		H[1] += b;
		H[2] += c;
		H[3] += d;
		H[4] += e;
	}

	/* Store binary digest in supplied buffer */
	if (digest)
	{
		for (idx = 0; idx < 5; idx++)
		{
			digest[idx * 4 + 0] = (uint8_t)(H[idx] >> 24);
			digest[idx * 4 + 1] = (uint8_t)(H[idx] >> 16);
			digest[idx * 4 + 2] = (uint8_t)(H[idx] >> 8);
			digest[idx * 4 + 3] = (uint8_t)(H[idx]);
		}
	}

	/* Store hex version of digest in supplied buffer */
	if (hexdigest)
	{
		simple_sprintf(hexdigest, "%08x%08x%08x%08x%08x",
			H[0], H[1], H[2], H[3], H[4]);
	}

	return 0;
}


__device__ int strLen(char* str) {
    int len = -1;
    while (str[len + 1] != NULL)
        len++;
    return len + 1;
}

__device__ void copy_str(char* dest, char* src) {
	int i = 0;
	while (src[i] != '\0') {
		dest[i] = src[i];
		i++;
	}
	dest[i] = '\0';
}


__device__ int str_cmp(char string1[], char string2[])
{
	for (int i = 0; ; i++)
	{
		if (string1[i] != string2[i])
		{
			return string1[i] < string2[i] ? -1 : 1;
		}

		if (string1[i] == '\0')
		{
			return 0;
		}
	}
}


__global__  void sha_find() {
	hiprandState_t state;
	char buf[20];

	copy_str(buf, word);

	/* we have to initialize the state */
	hiprand_init(0, /* the seed controls the sequence of random values that are produced */
		blockIdx.x * blockDim.x + threadIdx.x, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	while (found == 0 && strLen(buf) < 20)
	{
		uint8_t digest[20]; char hexdigest[41];
		int n = strLen(buf);
		buf[n] = char(hiprand(&state) % 127);
		buf[++n] = '\0';
		//printf("Hashing: %s\n", buf);
		sha1digest(digest, hexdigest, (uint8_t*)buf, strLen(buf));
		if (str_cmp(hexdigest, hash) == 0) {
			found = 1;
			//printf("The word is: %s", buf);
			copy_str(result, buf);
		}
		
	}
	/* hiprand works like rand - except that it takes a state as a parameter */
	
	//printf("Started\n");
	//printf("Word is %s\n", word);
	//printf("Rand: %d \n", result);
	/*uint8_t digest[20]; char hexdigest[41];
	sha1digest(digest, hexdigest, (uint8_t*)word, strLen(word));
	printf("%s\n", hexdigest);*/
}

// Helper function for using CUDA to add vectors in parallel.
void runCuda()
{
	double timee;
	struct timeb start, end;
	ftime(&start);
	while (found == 0)
	{
		sha_find << <1, 100 >> > ();
		hipDeviceSynchronize();
	};
	ftime(&end);
	timee = end.time - start.time + ((double)end.millitm - (double)start.millitm) / 1000.0;
	printf("The word is: %s\n", result);
	printf("Duration = % .2lf\n", timee);
	
}

int main()
{
    // Add vectors in parallel.
    runCuda();
 
    return 0;
}
